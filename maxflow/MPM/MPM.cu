#include "hip/hip_runtime.h"
#include "MPM.h"
#include "../bfs/bfs.h"

//Implementations of MPM functions members
#include "get_subgraph.cu"
#include "push_pull.cu"
#include "prune.cu"

#include <time.h>

#include <hip/hip_runtime_api.h>
#include "nvToolsExt.h"

#include "../config.h"

#define GPUID 0 
#define N_BLOCKS_MAX 65535
#define N_THREADS 512


MPM::MPM(csr_graph& _g) : g(_g) {
	//TODO reduce number of mallocs
	q_bfs  = (int*)my_malloc((g.n+1) * sizeof(int)); 
	h  = (int*)my_malloc((g.n) * sizeof(int)); 

	node_mask = (char*)my_malloc(g.n * sizeof(char)); 	
	queue_mask = (char*)my_malloc(g.n * sizeof(char)); 	
	prune_mask = (char*)my_malloc(g.n * sizeof(char)); 	
	have_been_pruned = (char*)my_malloc(g.n * sizeof(char)); 	

	node_g_to_sg  	= (int*)my_malloc(g.n * sizeof(int)); //TODO reuse Bfs
	node_sg_to_g  	= (int*)my_malloc(g.n * sizeof(int));

	edge_mask  	= (char*)my_malloc(g.nnz * sizeof(char));
	edge_mask_orig  = (char*)my_malloc(g.nnz * sizeof(char));
	reverse_edge_map  = (int*)my_malloc(g.nnz * sizeof(int));

	hipMalloc(&d_total_flow, sizeof(flow_t));
	e = (flow_t*)my_malloc(g.n * sizeof(flow_t)); 	

	//buffer for degree_in and degree_out
	degree 	= (flow_t*)my_malloc((2 * g.n) * sizeof(flow_t)); 	

	bfs_offsets 	= (int*)my_malloc((g.n+1) * sizeof(int)); 	
	sg_level_offsets 	= (int*)my_malloc((g.n+1) * sizeof(int)); 	

	hipMalloc(&d_nsg, sizeof(int));	

	hipHostMalloc(&d_node_to_push, sizeof(int));
	hipHostMalloc(&d_flow_to_push, sizeof(flow_t));

	hipStreamCreate(&st1);
	hipStreamCreate(&st2);

	hipMemset(d_total_flow, 0, sizeof(flow_t));	
	hipMemset(e, 0, sizeof(flow_t) * g.n);
	hipMemset(prune_mask, 0, sizeof(char) * g.n);


	buf1  = (int*)my_malloc((g.n+1) * sizeof(int)); 
	buf2  = (int*)my_malloc((g.n+1) * sizeof(int)); 

	sg_in.resize(g.n, g.nnz);
	sg_out.resize(g.n, g.nnz);

	cf = g.vals_cap; //TODO alloc and copy

	//CUB memory
	//Device Reduce

	hipMalloc(&d_ppd, sizeof(post_prune_data));

	hipcub::DeviceReduce::ArgMin(d_min_reduce, min_reduce_size, degree, &d_ppd->d_min, 2*g.n);
	hipMalloc(&d_min_reduce, min_reduce_size);
	
	//Partition (get subgraph)
	hipcub::DevicePartition::Flagged(d_storage_partition, size_storage_partition, buf1, queue_mask, buf2, d_nsg, g.n);
	hipMalloc(&d_storage_partition, size_storage_partition);
	
	//Exclusive sum (get subgraph)
	hipcub::DeviceScan::ExclusiveSum(d_storage_exclusive_sum, size_storage_exclusive_sum, buf1, buf2, g.n);
	hipMalloc(&d_storage_exclusive_sum, size_storage_exclusive_sum);
	

	//Building reverse edge map
	for(int u=0; u != g.n; ++u) {
		for (int i = g.row_offsets[u]; i < g.row_offsets[u+1]; ++i) {
			int v = g.col_indices[i];
			int uv = i;
			int vu = g.edge(v,u); 
			reverse_edge_map[uv] = vu;
		}
	}
	memFetch();
	hipDeviceSynchronize();
}

__global__ void setup_mask_unsaturated_kernel(int num_edges, char *mask, flow_t *cf)
{
	for(int u= threadIdx.x + blockIdx.x * blockDim.x;
		u < num_edges;
		u += blockDim.x * gridDim.x) 
		mask[u] = (cf[u] > 0);
}



bool setup_mask_unsaturated(int num_edges, char *mask, flow_t *cf) {
	setup_mask_unsaturated_kernel<<<min((num_edges + N_THREADS)/N_THREADS, N_BLOCKS_MAX), N_THREADS>>>(num_edges, mask, cf);
	return true;
}

//Main algorithm loop
flow_t MPM::maxflow(int _s, int _t, float *elapsed_time) {
	s = _s;
	t = _t;

	//TODO create cf
	setup_mask_unsaturated(g.nnz, edge_mask_orig, cf);

	
		int nsg; //number of nodes in subgraphh

	hipDeviceSynchronize();
	
	struct timespec start, end;
	clock_gettime(CLOCK_MONOTONIC, &start);
	hipProfilerStart();
	while(bfs(g.row_offsets, g.col_indices, g.n, g.nnz, s, t, q_bfs, h, BFS_MARK_DEPTH, edge_mask_orig, bfs_offsets)) {
		hipDeviceSynchronize();
		hipMemcpy(&ht, &h[t], sizeof(int), hipMemcpyDeviceToHost);
		init_level_graph(nsg);
		hipDeviceSynchronize();
		
		nvtxRangePushA("saturate_subgraph");	
		//Find node to push - usually done end of prune, but the first need to be done here
		hipcub::DeviceReduce::ArgMin(d_min_reduce, min_reduce_size, degree_in+1, &(d_ppd->d_min), 2*(sg_in.n-1), st1);

		hipMemcpy(&h_ppd, d_ppd, sizeof(post_prune_data), hipMemcpyDeviceToHost);
		do {
			push_and_pull();
			prune();
		} while(!h_ppd.s_t_pruned);		
		nvtxRangePop();	

	}

	flow_t h_total_flow;
	hipMemcpy(&h_total_flow, d_total_flow, sizeof(flow_t), hipMemcpyDeviceToHost);

	hipProfilerStop();
	clock_gettime(CLOCK_MONOTONIC, &end);
	*elapsed_time = (end.tv_sec - start.tv_sec) + (end.tv_nsec - start.tv_nsec) * 1e-9;

	return h_total_flow;
}

void MPM::memFetch() {
	hipMemPrefetchAsync(q_bfs, g.n * sizeof(int), 0, st1); 
	hipMemPrefetchAsync(h, (g.n) * sizeof(int), 0, st1); 
	
	hipMemPrefetchAsync(node_mask, g.n * sizeof(char), 0, st1); 	
	hipMemPrefetchAsync(queue_mask, g.n * sizeof(char), 0, st1); 	
	hipMemPrefetchAsync(prune_mask, g.n * sizeof(char), 0, st1); 	
	hipMemPrefetchAsync(have_been_pruned, g.n * sizeof(char), 0, st1); 	

	hipMemPrefetchAsync(node_g_to_sg, g.n * sizeof(int), 0, st1); //TODO reuse Bfs
	hipMemPrefetchAsync(node_sg_to_g, g.n * sizeof(int), 0, st1);
	
	hipMemPrefetchAsync(edge_mask, g.nnz * sizeof(char), 0, st1);
	hipMemPrefetchAsync(edge_mask_orig, g.nnz * sizeof(char), 0, st1);
	hipMemPrefetchAsync(reverse_edge_map, g.nnz * sizeof(int), 0, st1);
	
	hipMemPrefetchAsync(e, g.n * sizeof(flow_t), 0, st1); 	
	
	hipMemPrefetchAsync(bfs_offsets, (g.n+1) * sizeof(int), 0, st1); 	
	hipMemPrefetchAsync(sg_level_offsets, (g.n+1) * sizeof(int), 0, st1); 	
	
	hipMemPrefetchAsync(buf1, (g.n+1) * sizeof(int), 0, st1); 	
	hipMemPrefetchAsync(buf2, (g.n+1) * sizeof(int), 0, st1); 	
	
	hipMemPrefetchAsync(g.row_offsets, g.n * sizeof(int), 0, st1); 	
	hipMemPrefetchAsync(g.col_indices, g.nnz * sizeof(int), 0, st1); 	
	hipMemPrefetchAsync(cf, g.nnz * sizeof(flow_t), 0, st1); 	
}

MPM::~MPM() {
	//TODO free on host

} 
